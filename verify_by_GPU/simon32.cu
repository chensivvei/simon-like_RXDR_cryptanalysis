#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdint>
#include <random>
#include "common.h"


#define ROTL(x, n) ((((uint16_t)(x)<<(n)) | (uint16_t)(x)>>(16-(n))))
#define ROTR(x, n) ((((uint16_t)(x)>>(n)) | (uint16_t)(x)<<(16-(n))))


__global__ void encrypt(uint32_t *data1) {
    uint32_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    uint16_t right = tid;
    uint16_t left = tid >> 16;
    uint16_t tmp;
    uint32_t key[11] = {0x94f6, 0x3b02, 0xb740, 0x1a89, 0xff71, 0x806, 0x7ef0, 0xed8c, 0x5fe0, 0xb870, 0x6ce};
    for (int r = 0; r < 11; r++) {
        tmp = left;
        left = (ROTL(left, 8) & ROTL(left, 1)) ^ ROTL(left, 2) ^ right ^ key[r];
        right = tmp;
    }
    data1[tid] = ((uint32_t) left << 16) ^ right;
}

__global__ void encrypt2(uint32_t *data2) {
    uint32_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    uint16_t right;
    uint16_t left;
    uint16_t tmp;

    uint32_t key2[11] = {0x29eb, 0x7604, 0x6e81, 0x3512, 0xfee3, 0x100a, 0x5de6, 0x651f, 0x3e3, 0xcca1, 0x50d5};
    right = tid;
    right = ROTL(right, 1) ^ 0x0006;
    left = tid >> 16;
    left = ROTL(left, 1);
    for (int r = 0; r < 11; r++) {
        tmp = left;
        left = (ROTL(left, 8) & ROTL(left, 1)) ^ ROTL(left, 2) ^ right ^ key2[r];
        right = tmp;
    }
    data2[tid] = ((uint32_t) left << 16) ^ right;
}

__global__ void define(const uint32_t *data1, const uint32_t *data2, uint32_t *count) {
    uint64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t diff;
    for (uint64_t i = 0; i < (1 << 7); i++) {
        if (tid > i) {
            diff = data1[tid] ^ data1[i];
            if (diff == 0x20220008 && ((data2[tid] ^ data2[i]) == 0x40440010)) {
                atomicAdd(count, 1);
            }
        }
    }
}

__global__ void define11(const uint32_t *data1, const uint32_t *data2, uint32_t *count) {
    uint64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t diff;
    for (uint64_t i = 0; i < (1 << 21); i++) {
        if (tid > i) {
            diff = data1[tid] ^ data1[i];
            if (diff == 0x20220008 && ((data2[tid] ^ data2[i]) == 0x40440010)) {
                atomicAdd(count, 1);
            }
        }
    }
}

__global__ void define12(const uint32_t *data1, const uint32_t *data2, uint32_t *count) {
    uint64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t diff;
    for (uint64_t i = 0; i < (1 << 10); i++) {
        if (tid > i) {
            diff = data1[tid] ^ data1[i];
            if (diff == 0x20220008 && ((data2[tid] ^ data2[i]) == 0x40440010)) {
                atomicAdd(count, 1);
            }
        }
    }
}

#define KEYSIZE 65536
#define ROUND 16

__global__ void define3(uint32_t *count, uint16_t *keylist) {
    uint32_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    uint16_t right;
    uint16_t left;
    uint16_t tmp;

    //decrypt c1
    for (int i = 0; i < KEYSIZE; i++) {
        int key_index = i * ROUND * 2;
        uint16_t *key = keylist + key_index;
        uint16_t *key2 = keylist + key_index + ROUND;

        right = tid >> 16;
        left = tid;
        for (int r = 0; r < ROUND; r++) {
            tmp = left;
            left = (ROTL(left, 8) & ROTL(left, 1)) ^ ROTL(left, 2) ^ right ^ key[ROUND - 1 - r];
            right = tmp;
        }
        uint32_t p1 = ((uint32_t) right << 16) ^ left;

        //decrypt c3

        uint32_t c3 = tid ^ 0x20220008;

        right = c3 >> 16;
        left = c3;
        for (int r = 0; r < ROUND; r++) {
            tmp = left;
            left = (ROTL(left, 8) & ROTL(left, 1)) ^ ROTL(left, 2) ^ right ^ key[ROUND - 1 - r];
            right = tmp;
        }
        uint32_t p3 = ((uint32_t) right << 16) ^ left;

        right = p1;
        left = p1 >> 16;

        uint32_t p2 = ((uint32_t) ROTL(left, 1) << 16) ^ ROTL(right, 1) ^ 0x00000006;

        right = p3;
        left = p3 >> 16;
        uint32_t p4 = ((uint32_t) ROTL(left, 1) << 16) ^ ROTL(right, 1) ^ 0x00000006;

        //encrypt p2
        right = p2;
        left = p2 >> 16;
        for (int r = 0; r < ROUND; r++) {
            tmp = left;
            left = (ROTL(left, 8) & ROTL(left, 1)) ^ ROTL(left, 2) ^ right ^ key2[r];
            right = tmp;
        }
        uint32_t c2 = ((uint32_t) left << 16) ^ right;

        //encrypt p4
        right = p4;
        left = p4 >> 16;
        for (int r = 0; r < ROUND; r++) {
            tmp = left;
            left = (ROTL(left, 8) & ROTL(left, 1)) ^ ROTL(left, 2) ^ right ^ key2[r];
            right = tmp;
        }
        uint32_t c4 = ((uint32_t) left << 16) ^ right;

        if ((c4 ^ c2) == 0x40440010) {
            atomicAdd(count + i, 1);
//            if (tid < 0xffffff){
//                printf("c1 %#x c2 %#x c3 %#x c4 %#x p1 %#x p2 %#x p3 %#x p4 %#x\n",tid, c2,c3,c4,p1,p2,p3,p4);
//            }

        }

    }
}

void keyschedule(uint16_t *key, int R) {
    uint16_t c;
    for (int i = 0; i < R - 4; i++) {
        int r = i + 1;
        if ((r == 1) || (r == 2) || (r == 3) || (r == 4)
            || (r == 5) || (r == 7) || (r == 11) ||
            (r == 14) || (r == 16) || (r == 18) ||
            (r == 19) || (r == 24) || (r == 25) ||
            (r == 26)) {
            c = 0xfffd;
        } else {
            c = 0xfffc;
        }
        key[i + 4] = (ROTR(key[i + 3], 3) ^ key[i + 1]) ^ ROTR((ROTR(key[i + 3], 3) ^ key[i + 1]), 1) ^ c ^ key[i];
    }
}

void init_key(uint16_t *key) {
    using std::default_random_engine;
    default_random_engine e;
    e.seed(time(NULL));
    for (int i = 0; i < KEYSIZE; i++) {
        int index = i * ROUND * 2;
        key[index] = e();
        key[index + 1] = e();
        key[index + 2] = e();
        key[index + 3] = e();
        keyschedule(key + index, ROUND);
        key[index + ROUND] = ROTL(key[index], 1) ^ 0x0006;
        key[index + ROUND + 1] = ROTL(key[index + 1], 1);
        key[index + ROUND + 2] = ROTL(key[index + 2], 1);
        key[index + ROUND + 3] = ROTL(key[index + 3], 1);
        keyschedule(key + index + ROUND, ROUND);
    }
}


int main() {
//    printf("%#x \n %#x\n", ROTL(0x2022,1), ROTL(0x0008,1));//0x4044   0x10

    hipSetDevice(3);

    size_t nWords = 1ULL << 32;

    uint32_t *count;
    uint16_t *key;

    CHECK(hipMallocManaged((uint32_t **) &count, KEYSIZE * sizeof(uint32_t)));
    CHECK(hipMallocManaged((uint32_t **) &key, KEYSIZE * sizeof(uint16_t) * ROUND * 2));

    memset(count, 0, KEYSIZE * sizeof(uint32_t));

    init_key(key);

//    printf("key: ");
//    for (int j = 0; j < 11; j++) {
//        printf("%#x ", key[j]);
//    }
//    printf("\n");
//    printf("key2: ");
//    for (int j = 0; j < 11; j++) {
//        printf("%#x ", key[11 + j]);
//    }
//    printf("\n");

    int blockx = 1024;
    dim3 block = blockx;
    dim3 grid = nWords / blockx;

    define3<<<grid, block>>>(count, key);

    CHECK(hipDeviceSynchronize());

//    for (int i = 0; i < KEYSIZE; i++) {
//        for (int j = 0; j < 11; j++) {
//            printf("%#x ",key[j + i * 22]);
//        }
//        printf("\n");
//        for (int j = 0; j < 11; j++) {
//            printf("%#x ",key[j + 11 + i * 22]);
//        }
//        printf("\n");
//        printf("\n");
//    }


    for (int i = 0; i < KEYSIZE; i++) {
        printf("count %d is %d\n",i,count[i]);
    }

    uint64_t problity = 0;
    for (int i = 0; i < KEYSIZE; i++) {
        problity += count[i];
    }
    printf("\nThe probability is %f\n",(double)problity/KEYSIZE);

//    hipFree(d_data1);
//    hipFree(d_data2);
//    free(h_data1);
//    free(h_data2);
    return 0;
}
